#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

hipArray *d_volumeArray_test = 0;
hipTextureObject_t texObject_test;

typedef short VolumeType;

__global__ void transformKernel(float* output,
                                hipTextureObject_t texObj,
                                int width, int height) 
{
	const int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	const int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    output[y * width + x] = tex3D<float>(texObj, 1.0*x/width, 0, 0)*32768;
}

extern "C" 
void cu_test_3d( VolumeType* h_volumeData, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray_test, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(
        h_volumeData,
        volumeSize.width * sizeof(VolumeType),
        volumeSize.width, 
        volumeSize.height
    );
    copyParams.dstArray = d_volumeArray_test;
    copyParams.extent = volumeSize;
    copyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_volumeArray_test;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    // texDescr.filterMode = hipFilterModePoint;
    texDescr.filterMode = hipFilterModeLinear;

    texDescr.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;

    // texDescr.readMode = hipReadModeElementType;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(
        hipCreateTextureObject(&texObject_test, &texRes, &texDescr, NULL)
    );

    float* result_arr;
    hipMalloc(&result_arr, volumeSize.width * volumeSize.height * sizeof(float));

    dim3 blockSize(16, 16);
	dim3 gridSize( (volumeSize.width-1)/blockSize.x+1, (volumeSize.height-1)/blockSize.y+1 );
    transformKernel<<<gridSize, blockSize>>>(result_arr, texObject_test, volumeSize.width, volumeSize.height);

    float* pOut = (float*)malloc(volumeSize.width * volumeSize.height * sizeof(float));

    hipMemcpy( pOut, result_arr, volumeSize.width * volumeSize.height * sizeof(VolumeType), hipMemcpyDeviceToHost );

    for (int i=0; i<volumeSize.width; i++){
        printf("%.0f ", pOut[i]);
    }
    printf("\n\n");
    for (int i=volumeSize.width; i<volumeSize.width*2; i++){
        printf("%.2f ", pOut[i]);
    }

    free(pOut);
}

hipArray *d_transferFuncArray_test = 0;
hipTextureObject_t transferTex_test;


__global__ void transformKernel_1d(
    float* output,
    hipTextureObject_t texObj,
    int nLen
) 
{
	const int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    float4 c = tex1D<float4>(texObj, 1.0*x/nLen);
    output[4*x] = c.x;
    output[4*x+1] = c.y;
    output[4*x+2] = c.z;
    output[4*x+3] = c.w;
}


hipResourceDesc texRes;
hipTextureDesc texDescr;

extern "C"
void cu_init_test_1d()
{
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;

    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;

    // checkCudaErrors(
    //     hipCreateTextureObject(&transferTex_test, &texRes, &texDescr, NULL)
    // );
}


extern "C" 
void cu_test_1d( float* pTransferFunc, int nLenTransferFunc )
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    checkCudaErrors(hipMallocArray( &d_transferFuncArray_test, &channelDesc, nLenTransferFunc, 1));
    checkCudaErrors(
        hipMemcpy2DToArray(
            d_transferFuncArray_test, 
            0, 
            0, 
            pTransferFunc,
            0, 
            nLenTransferFunc*sizeof(float4), 
            1,
            hipMemcpyHostToDevice
        )
    );

    // hipResourceDesc texRes;
    // memset(&texRes, 0, sizeof(hipResourceDesc));

    // texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_transferFuncArray_test;

    // hipTextureDesc texDescr;
    // memset(&texDescr, 0, sizeof(hipTextureDesc));

    // texDescr.normalizedCoords = true;
    // texDescr.filterMode = hipFilterModeLinear;
    // texDescr.addressMode[0] = hipAddressModeClamp;
    // texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(
        hipCreateTextureObject(&transferTex_test, &texRes, &texDescr, NULL)
    );

    float* result_arr;
    hipMalloc(&result_arr, nLenTransferFunc * 4 * sizeof(float));

    dim3 blockSize(16);
	dim3 gridSize( (nLenTransferFunc-1)/blockSize.x+1 );
    transformKernel_1d<<<gridSize, blockSize>>>(result_arr, transferTex_test, nLenTransferFunc);

    float* pOut = (float*)malloc(nLenTransferFunc * 4 * sizeof(float));

    hipMemcpy( pOut, result_arr, nLenTransferFunc * 4 * sizeof(float), hipMemcpyDeviceToHost );

    for (int i=0; i<nLenTransferFunc; i++){
        printf("%.2f %.2f %.2f %.2f\n", pOut[4*i], pOut[4*i+1], pOut[4*i+2], pOut[4*i+3]);
    }
    
    free(pOut);
}

